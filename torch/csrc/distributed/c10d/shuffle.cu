#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

constexpr int64_t BYTES_PER_THREAD = 16;
constexpr int64_t MAX_NUM_THREADS = 1024;
constexpr int64_t MIN_NUM_THREADS = 128;
constexpr int64_t WARP_SIZE = 32;

template <typename T>
__device__ inline void streamLoad128(uint4& val, const T* addr) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  unsigned long long int low, high;
  asm("ld.global.nc.v2.u64 {%0, %1}, [%2];"
      : "=l"(low), "=l"(high)
      : "l"(addr));
  reinterpret_cast<unsigned long long int*>(&val)[0] = low;
  reinterpret_cast<unsigned long long int*>(&val)[1] = high;
#endif
}

template <typename T>
__device__ inline void streamStore128(T* addr, const uint4& val) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  unsigned long long int low, high;
  low = reinterpret_cast<const unsigned long long int*>(&val)[0];
  high = reinterpret_cast<const unsigned long long int*>(&val)[1];
  asm("st.global.cs.v2.u64 [%0], {%1, %2};" : : "l"(addr), "l"(low), "l"(high));
#endif
}

static __host__ __device__ inline int64_t divUp(int64_t a, int64_t b) {
  return (a + b - 1) / b;
}

static __device__ inline bool isAligned(const void* ptr, size_t alignment) {
  uintptr_t addr = reinterpret_cast<uintptr_t>(ptr);
  return addr % alignment == 0;
}

template <typename T>
static __global__ void fsdpAllGatherCopyOutKernel(
    T** paramPtrs,
    T* allGatherResPtr,
    int64_t numel,
    int64_t* blockOffsetToParamIdx,
    int64_t* blockCumSums,
    int64_t* shardDimCumSums,
    int64_t numParams,
    int64_t shardDimSum,
    int64_t blockDimSum,
    int64_t ranksPerBlock,
    int64_t worldSize) {
  constexpr int64_t numelPerThread = BYTES_PER_THREAD / sizeof(T);

  const int64_t blockOffset = blockIdx.x % blockDimSum;
  const int64_t paramIdx = blockOffsetToParamIdx[blockOffset];

  for (int64_t rank = blockIdx.x / blockDimSum; rank < worldSize;
       rank += worldSize / ranksPerBlock) {
    const int64_t shardBlockCount =
        blockCumSums[paramIdx + 1] - blockCumSums[paramIdx];
    const int64_t groupSize = shardBlockCount * blockDim.x;
    const int64_t localTid =
        (blockOffset - blockCumSums[paramIdx]) * blockDim.x + threadIdx.x;

    const int64_t shardBegin = shardDimCumSums[paramIdx];
    const int64_t shardEnd = shardDimCumSums[paramIdx + 1];
    const int64_t shardLen = shardEnd - shardBegin;
    const int64_t srcOff = rank * shardDimSum + shardBegin;
    const int64_t dstOff = rank * shardLen;

    const T* srcPtr = allGatherResPtr + srcOff;
    T* dstPtr = &paramPtrs[paramIdx][dstOff];

    const int64_t alignOff =
        divUp(dstOff, numelPerThread) * numelPerThread - dstOff;
    const int64_t begin = alignOff + localTid * numelPerThread;
    const int64_t end =
        alignOff + (shardLen - alignOff) / numelPerThread * numelPerThread;
    const int64_t stride = groupSize * numelPerThread;

    for (size_t i = begin; i < end; i += stride) {
      uint4 val;
      if (isAligned(srcPtr + i, 128)) {
        streamLoad128(val, srcPtr + i);
      } else {
        for (size_t j = 0; j < numelPerThread; ++j) {
          reinterpret_cast<T*>(&val)[j] = srcPtr[i + j];
        }
      }
      streamStore128(&dstPtr[i], val);
    }
    if (localTid < alignOff && localTid < shardLen) {
      dstPtr[localTid] = srcPtr[localTid];
    }
    if (end + localTid < shardLen) {
      dstPtr[end + localTid] = srcPtr[end + localTid];
    }
  }
}

std::pair<at::Tensor, std::vector<int64_t*>> pack(
    std::vector<std::vector<int64_t>> vecs) {
  int64_t numel = 0;
  for (const auto& vec : vecs) {
    numel += vec.size();
  }

  auto packed = at::empty(
      {numel}, at::TensorOptions().dtype(at::kLong).pinned_memory(true));
  size_t offset = 0;
  for (const auto& vec : vecs) {
    memcpy(
        packed.data_ptr<int64_t>() + offset,
        vec.data(),
        sizeof(int64_t) * vec.size());
    offset += vec.size();
  }
  packed = packed.cuda();

  std::vector<int64_t*> ptrs;
  offset = 0;
  for (const auto& vec : vecs) {
    ptrs.push_back(packed.data_ptr<int64_t>() + offset);
    offset += vec.size();
  }
  return std::make_pair(packed, ptrs);
}

void fsdpAllGatherCopyOut(
    std::vector<at::Tensor> params,
    at::Tensor allGatherRes,
    int64_t worldSize,
    int64_t maxBlocksPerShard) {
  const auto device = allGatherRes.device();
  const auto scalarType = allGatherRes.scalar_type();

  TORCH_CHECK(allGatherRes.is_cuda());
  TORCH_CHECK(allGatherRes.is_non_overlapping_and_dense());
  TORCH_CHECK(allGatherRes.numel() % worldSize == 0);

  std::vector<int64_t> paramPtrs;
  std::vector<int64_t> dimCumSums{0};
  for (size_t i = 0; i < params.size(); ++i) {
    const auto& param = params[i];
    TORCH_CHECK(param.is_non_overlapping_and_dense());
    TORCH_CHECK(param.device() == device);
    TORCH_CHECK(param.scalar_type() == scalarType);
    // All params are expected to be aligned at worldSize.
    // But not neccessarily worldSize * numelPerThread.
    TORCH_CHECK(param.numel() % worldSize == 0);
    paramPtrs.push_back(reinterpret_cast<int64_t>(param.data_ptr()));
    dimCumSums.push_back(dimCumSums[i] + param.numel() / worldSize);
  }

  TORCH_CHECK(
      dimCumSums.back() * worldSize == allGatherRes.numel(),
      "allGatherRes and params must contain the same number of elements.");

  // To balance the throughput larger shards and waste on smaller shards,
  // determine the block size with the average shard length.
  const int64_t numelPerThread = BYTES_PER_THREAD / params[0].element_size();
  const int64_t avgShardLen = allGatherRes.numel() / worldSize / params.size();
  int64_t blockSize = divUp(avgShardLen, numelPerThread);
  blockSize = divUp(blockSize, WARP_SIZE) * WARP_SIZE;
  blockSize = std::min(std::max(blockSize, MIN_NUM_THREADS), MAX_NUM_THREADS);

  // TODO: if the numBlocks produced at this stage far exceeds maxActiveBlocks,
  // we should increase the iter factor here as well.
  std::vector<int64_t> blockOffsetToParamIdx;
  std::vector<int64_t> blockCumSums{0};
  for (int64_t paramIdx = 0; paramIdx < static_cast<int64_t>(params.size());
       ++paramIdx) {
    const int64_t shardNumel = params[paramIdx].numel() / worldSize;
    int64_t numBlocks = divUp(shardNumel, blockSize * numelPerThread);
    numBlocks = std::min(numBlocks, maxBlocksPerShard);
    blockOffsetToParamIdx.insert(
        blockOffsetToParamIdx.end(), numBlocks, paramIdx);
    blockCumSums.push_back(blockCumSums.back() + numBlocks);
  }
  const auto numBlocks = blockCumSums.back();

  auto packed =
      pack({paramPtrs, blockOffsetToParamIdx, blockCumSums, dimCumSums});

  // TODO: this is only for A100
  constexpr int64_t maxActiveBlocks = 32 * 108;
  int64_t ranksPerBlock = 1;
  while (numBlocks * (worldSize / ranksPerBlock) < maxActiveBlocks &&
         ranksPerBlock < worldSize) {
    ++ranksPerBlock;
  }

  dim3 blocks(numBlocks * (worldSize / ranksPerBlock), 1, 1);
  dim3 threads(blockSize, 1, 1);

  LOG(INFO) << "blocks: " << blocks.x << ", threads: " << threads.x;
  LOG(INFO) << "avgShardLen: " << avgShardLen
            << ", ranksPerBlock: " << ranksPerBlock;

  AT_DISPATCH_ALL_TYPES_AND(
      at::ScalarType::BFloat16, scalarType, "fsdp_all_gather_copy_out", [&] {
        fsdpAllGatherCopyOutKernel<scalar_t>
            <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                reinterpret_cast<scalar_t**>(packed.second[0]),
                allGatherRes.data_ptr<scalar_t>(),
                allGatherRes.numel(),
                /*blockOffsetToParamIdx=*/packed.second[1],
                /*blockCumSums=*/packed.second[2],
                /*shardDimCumSums=*/packed.second[3],
                params.size(),
                dimCumSums.back(),
                blockCumSums.back(),
                ranksPerBlock,
                worldSize);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}
